#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>
#include <string>
#include <time.h>

using namespace std;

#include "matrix.h"
#include "utils.h"
#include "crbm.h"

typedef MatrixGpu Mat;

int main(int argc, char** argv)
{
    if(argc != 7 && argc != 8)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " input-vector-file input-weight-file hidden-size learning-speed iter batch [cudadevice-id]" << endl;
        exit(1);
    }

    if(argc > 7)
    {
        int device = atoi(argv[7]);
        cout << "Device ID: " << device << endl;
        hipSetDevice(device);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    cout << "cublas init ..." << flush;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    cout << " done" << endl;

    int hidden = atoi(argv[3]);
    float lSpeed = atof(argv[4]);
    float iterations = atof(argv[5]);
    int batchSize = atoi(argv[6]);

    MatrixCpu *xCpu = new MatrixCpu();
    loadMatrix(*xCpu, argv[1]);
    Mat xx = *xCpu;

    delete xCpu;
    xCpu = new MatrixCpu();


    CRBM::CRBMLayer *abc = NULL;

    Timer timer;
    if(string(argv[2]) != "-")
    {
        abc = new CRBM::CRBMLayer();
        abc->Load(string(argv[2]));
    }
    else
    {
//#define TEST
#ifdef TEST
        //image-size
        int im_x = 3;
        int im_y = 4;
        int im_z = 2;
    
        //convolution-size
        int im_cx = 2;
        int im_cy = 2;
    
        //stride-size
        int im_stridex = 1;
        int im_stridey = 1;
#else
        //image-size
        int im_x = 200;
        int im_y = 200;
        int im_z = 3;
    
        //convolution-size
        int im_cx = 10;
        int im_cy = 10;
    
        //stride-size
        int im_stridex = 5;
        int im_stridey = 5;
#endif

        cout << "Creating RBM-layer ... " << flush;
        abc = new CRBM::CRBMLayer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, hidden);
        abc->setLearningSpeed(lSpeed);
        timer.tac("done ");

    }


    timer.tic();
    abc->LearnBatch(xx, iterations);
    timer.tac("learning duration: ");

    Mat transformed;
    abc->Transform(xx, transformed);

    saveMatrix(transformed, string(argv[1]) + ".transformed");
    abc->Save(string(argv[1]) + ".rbm");

    return 0;
}
