#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>
#include <string>
#include <time.h>
#include <csignal>

using namespace std;

#include "matrix.h"
#include "utils.h"
#include "crbm.h"

typedef MatrixGpu Mat;

CRBM::CRBMLayer *abc = NULL;

void signalHandler(int signum)
{
    if(abc != NULL)
    {
        cout << endl;
        cout << "!!! Forcing RBM to interrupt learning ...            !!!" << endl;
        cout << "!!! repeated CTRL+C will stop program without saving !!!" << endl;
        cout << endl;

        abc->SignalStop();

        //clear handler
        signal(SIGINT, SIG_DFL);
    }
    else
    {
        exit(signum);
    }
}


int main(int argc, char** argv)
{
    if(argc != 4 && argc != 5)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " setting-file model-file input-vector-file [cudadevice-id]" << endl;
        cout << "\tmodel-file can be \"-\" for random-model initialization." << endl;
        exit(1);
    }

    if(argc > 4)
    {
        int device = atoi(argv[4]);
        cout << "Device ID: " << device << endl;
        hipSetDevice(device);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    cout << "cublas init ..." << flush;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    cout << " done" << endl;

    CRBM::CRBMLayerSetting setting;
    setting.loadFromFile(argv[1]);

    //register signal SIGINT and signal handler  
    signal(SIGINT, signalHandler);

    Timer timer;
    if(string(argv[2]) != "-")
    {
        cout << "Loading RBM-layer ... " << flush;
        abc = new CRBM::CRBMLayer(setting);
        abc->Load(string(argv[2]));

        //reset loaded setting
        abc->ResetSetting(setting);
    }
    else
    {
        cout << "Creating RBM-layer ... " << flush;
        abc = new CRBM::CRBMLayer(setting);
        timer.tac("  ... done in ");
    }

    MatrixCpu *xCpu = new MatrixCpu();
    loadMatrix(*xCpu, argv[3]);
    Mat xx = *xCpu;

    delete xCpu;
    xCpu = new MatrixCpu();


    timer.tic();
    abc->LearnAll(xx, string(argv[3]) + ".rbm");
    timer.tac("learning duration: ");

    if(abc->IsStopRequired())
    {
        cout << endl;
        for(int i = 3; i > 0; --i)
        {
            cout << "\rsave will be started in " << i << flush;
            sleep(1);
        }
        cout << "\rsave will be started now! " << endl;
    }

    abc->Save(string(argv[3]) + ".rbm");

    Mat transformed;
    abc->Transform(xx, transformed);
    saveMatrix(transformed, string(argv[3]) + ".transformed");

    Mat reconstructed;
    abc->Reconstruct(transformed, reconstructed);
    saveMatrix(reconstructed, string(argv[3]) + ".reconstructed");

    return 0;
}
