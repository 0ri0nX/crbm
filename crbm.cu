#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>
#include <string>
#include <time.h>

using namespace std;

/*
//#define CUDA

#ifdef CUDA
    //#include "deviceGPU.h"
    #include<hipblas.h>


#else
    #include "deviceCPU.h"
    typedef ComputingDevice::DeviceCPU dev;
#endif
*/

#include "matrix.h"

class Timer
{
    public:
        Timer(void)
        {
            tic();
        }

        void tic(void)
        {
            clock_gettime(CLOCK_MONOTONIC, &m_TimeSpec);
        }

        void tac(const string &inComment = "")
        {
            timespec ts;
            clock_gettime(CLOCK_MONOTONIC, &ts);

            float t = (ts.tv_sec - m_TimeSpec.tv_sec) + (ts.tv_nsec - m_TimeSpec.tv_nsec)/10e9;

            cout << inComment << t << " sec" << endl;
        }

    private:

        timespec m_TimeSpec;
};


using namespace YAMATH;

//class CRBMLayer
//{
//    //image-size
//    int m_x = 200;
//    int m_y = 200;
//    int m_z = 3;
//
//    //convolution-size
//    int m_cx = 10;
//    int m_cy = 10;
//
//    //stride-size
//    int m_stridex = 5;
//    int m_stridey = 5;
//
//    MatrixGpu m_Weights;
//
//    int pn = convolutionPatchesNumber(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey);
//    Mat x, xraw, y, x2, y2, dw1, dw2, err, lastW;
//    x = xx.Convolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey);
//
//    Mat w(x.getY(), hidden); //init weights
//
//};

void msgC(const char * inMsg, const MatrixCpu &x)
{
    int n = x.getX()*x.getY();
    if(n > 400)
    {
        cout << inMsg << ": " << x.getX() << " x " << x.getY()
             << "[ " << (x.getDataConst()[0]) << ", " << (x.getDataConst()[1]) << " ... " << (x.getDataConst()[n-2]) << ", " << (x.getDataConst()[n-1]) << " ]" << endl;
    }
    else if(n == 1)
    {
        cout  << inMsg << ":[" << x.getDataConst()[0] << "]" << flush;
    }
    else
    {
        cout  << inMsg << ":" << endl;
        x.Save(cout);
        cout << endl;
    }
}

void msgG(const char * inMsg, const MatrixGpu &inM)
{
    MatrixCpu x = inM;
    msgC(inMsg, x);
}

void loadMatrix(MatrixCpu &inM, const string& filename, bool inTransposed = false)
{
    cout << "loading [" << filename << "] ... " << endl;
    Timer t;
    ifstream f(filename.c_str());
    inM.Load(f, inTransposed);
    f.close();
    t.tac("   ... done in ");
    msgC(filename.c_str(), inM);
}

void saveMatrix(MatrixCpu &inM, const string &filename)
{
    cout << "saving [" << filename << "] ... " << endl;
    Timer t;
    ofstream f(filename.c_str());
    inM.Save(f);
    f.close();
    t.tac("   ... done in ");
    msgC(filename.c_str(), inM);
}

void saveGpuMatrix(MatrixGpu &inM, const string &filename)
{
    MatrixCpu resx = inM;
    saveMatrix(resx, filename);
}



void ms(char * inMsg, const MatrixGpu &inM)
{
    //msgG(inMsg, inM);
}


void testGpu(int x, int y)
{
    typedef MatrixGpu M;
    typedef MatrixCpu MC;
    
    cout << "GPU -----------" << endl;

    MC ac(x, y);
    for(int i = 0; i < ac.getX()*ac.getY(); ++i)
    {
        ac.getData()[i] = float(i);
    }
    M a = ac;
    msgG("a - init", a);

    //a = 11.0f;
    //msgG("a=11.0f", a);

    M b = a.AbsSum();
    msgG("b=a.AbsSum()", b);

    MC cc(y, 3);
    for(int i = 0; i < cc.getX()*cc.getY(); ++i)
    {
        cc.getData()[i] = 0.0f;
    }
    cc.getData()[0] = 1.0f;
    cc.getData()[y+1] = 1.0f;
    cc.getData()[2*y+2] = 1.0f;
    M c = cc;
    msgG("c", c);

    M d = Mult(a, c);
    msgG("d=a*c", d);

    
}

void testCpu(int x, int y)
{
    typedef MatrixCpu M;

    cout << "CPU -----------" << endl;
    
    M a(x, y);
    msgC("a - init", a);

    //a = 11.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        a.getData()[i] = 11;
    }
    msgC("a=11.0f", a);

    M b(1,1);
    float sum = 0.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        sum += a.getData()[i];
    }
    b.getData()[0] = sum;
    msgC("sum=a.AbsSum()", b);
}

//const float x1[] = {1.0f, 0.0f, 0.0f};
//const float t1[] = {1.0f, 0.0f};
//
//const float x2[] = {1.0f, 0.0f, 1.0f};
//const float t2[] = {1.0f, 0.0f};
//
//const float x3[] = {1.0f, 1.0f, 0.0f};
//const float t3[] = {1.0f, 0.0f};
//
//const float x4[] = {1.0f, 1.0f, 1.0f};
//const float t4[] = {0.0f, 1.0f};

typedef MatrixGpu Mat;

float computeError(Mat &inInp, Mat &inOut)
{
    Mat r2, r3;
    r2 = inInp - inOut;
    //msgG("in", inInp);
    //msgG("out", inOut);
    //msgG("r2", r2);
    r2 ^= 2.0f;
    r3 = r2.Sum();
    r3 *= 1.0f / inInp.getX();

    MatrixCpu rr = r3;

    msgG("abssum2", r3);

    return rr.getDataConst()[0];
}

int main(int argc, char** argv)
{
    if(argc != 7 && argc != 8)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " input-vector-file input-weight-file hidden-size learning-speed iter batch [cudadevice-id]" << endl;
        exit(1);
    }

    if(argc > 7)
    {
        int device = atoi(argv[7]);
        cout << "Device ID: " << device << endl;
        hipSetDevice(device);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    cout << "cublas init ..." << flush;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    cout << " done" << endl;

    int hidden = atoi(argv[3]);
    float lSpeed = atof(argv[4]);
    float iterations = atof(argv[5]);
    int batchSize = atoi(argv[6]);

    MatrixCpu *xCpu = new MatrixCpu();

    loadMatrix(*xCpu, argv[1]);

    int rows = xCpu->getX();
    int cols = xCpu->getY();

    Mat xx = *xCpu;
    msgG("loaded", xx);

    delete xCpu;
    xCpu = new MatrixCpu();

    if(string(argv[2]) != "-")
    {
        loadMatrix(*xCpu, string(argv[2]));
        msgC("w", *xCpu);
    }

//#define TEST
#ifdef TEST
    //image-size
    int im_x = 3;
    int im_y = 4;
    int im_z = 2;

    //convolution-size
    int im_cx = 2;
    int im_cy = 2;

    //stride-size
    int im_stridex = 1;
    int im_stridey = 1;
#else
    //image-size
    int im_x = 200;
    int im_y = 200;
    int im_z = 3;

    //convolution-size
    int im_cx = 10;
    int im_cy = 10;

    //stride-size
    int im_stridex = 5;
    int im_stridey = 5;
#endif

    Timer timer;

    int transX, transY;

    convolutionPatchesNumber(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, transX, transY);

    cout << "On image " << im_x << "x" << im_y << "x" << im_z << " applied convolution " << im_cx << "x" << im_cy << " with stride " << im_stridex << "x" << im_stridey << endl;
    cout << "It resulted into " << transX << "x" << transY << " patches." << endl;

    //int pn = transX*transY;

    Mat x, xraw, y, x2, y2, dw1, dw2, err, lastW;

    timer.tic();
    x = xx.Convolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey);
    timer.tac("Convolve: ");

    Mat w(x.getY(), hidden); //init weights
    cout << xCpu->getX() << ", " << xx.getX() << ", " << xCpu->getY() << ", " << hidden << endl;

    if(xCpu->getX() != w.getX() || xCpu->getY() != w.getY())
    {
        w.RandNormal(0.0f, 1.0f/(10*hidden));
        cout << "weight matrix randomized!" << endl;
    }
    else
    {
        w = *xCpu;
        cout << "weight matrix loaded!" << endl;
    }
    //msgG("w", w);
    delete xCpu;


    //w = 0.0f;
    ms("w", w);

    lastW = w;

    cout << endl;
    
    float minErr = FLT_MAX;
    int minIndex = 0;

    bool ONE_ROW = true;


    msgG("xxxxx", x);
    msgG("wwwww", w);

  
    for(int i = 0; i < iterations; ++i)
    {
        //Mat xraw = xx;
        //Mat xraw = xx.Sample(batchSize);

        //x = xraw.Convolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey);

        //cout << "x:" << x.getX() << ", y:" << x.getY() << endl;

        //saveGpuMatrix(x, string(argv[1]) + ".convolved");

        //Mat reverse;
        //reverse = x.DeConvolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, normalizer);

        //saveGpuMatrix(reverse, string(argv[1]) + ".reversed");
        //exit(1);

        y = Mult(x, w); // matrixwise -  y.shape = (dataA.x, weights.y) == (dataB.x, dataB.y)
        //msgG("y", y);
        //msgG("y=x*w", y);

        //y = y.Sigmoid();
        //msgG("y", y);

        x2 = Mult(y, w.T());
        //msgG("x2", x2);

        //x2 = x2.Sigmoid();
        //msgG("x2", x2);

        y2 = Mult(x2, w);
        //msgG("y2", y2);

        //y2 = y2.Sigmoid();
        //msgG("y2", y2);

        dw1 = Mult(x.T(), y);
        //msgG("dw1", dw1);
        dw2 = Mult(x2.T(), y2);
        //msgG("dw2", dw2);

        dw1 *= (lSpeed/x.getX());
        dw2 *= (lSpeed/x.getX());

        w = w + dw1;
        w = w - dw2;

        //lastW *= 0.00001;
        //w = w - lastW;

        lastW = w;
        //msgG("w", w);

        ms("w = w + dw", w);

        if(i % 50 == 0 || i+1 == iterations )
        {
            cout << i << ": ";
            float terr = computeError(x, x2);

            cout << "              " << flush;

            if(ONE_ROW)
            {
                cout << "              " << "\r" << flush;
            }
            else
            {
                cout << endl;
            }
            if(terr < minErr)
            {
                minErr = terr;
                minIndex = i;
            }
        }
    }
    cout << endl;

    MatrixCpu res = w;

    msgC("res", res);
    saveMatrix(res, string(argv[1]) + ".weights");

    y = Mult(x, w);
    //y = y.Sigmoid();
    //msgG("y - raw", y);
    //y.Reshape(xx.getX(), transX*transY*hidden);
    msgG("y", y);
    Mat yyy = y.TransformToUpperLayer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, xx.getX());
    msgG("trans(y)", yyy);
    Mat zzz = yyy.TransformFromUpperLayer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, xx.getX());
    msgG("retranst(trans(y))", zzz);

    //y.Reshape(hidden*xx.getX(), transX*transY);
    //msgG("reshaped(y)", y);
    //MatrixCpu resy = y;
    //saveMatrix(resy, string(argv[1]) + ".transformRaw");

    //exit(1);
    
    //Mat yy = y.T();
    //yy.MakeHardCopy();
    //msgG("transposed(y)", yy);
    //saveMatrix(resy, string(argv[1]) + ".transformRawTransposed");
    
    //saveMatrix(resy, string(argv[1]) + ".transform");
    exit(1);
    x2 = Mult(y, w.T());
    //x2 = x2.Sigmoid();
    Mat reverse, normalizer;
    timer.tic();
    normalizer = x.DeConvolveNormalizer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, xx.getX());
    timer.tac("DeConvolveNormalizer: ");
    timer.tic();
    reverse = x2.DeConvolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, normalizer);
    timer.tac("DeConvolve: ");
    MatrixCpu resx = reverse;
    saveMatrix(resx, string(argv[1]) + ".reconstruct");


    cout << "done" << endl << "Min. test error = " << minErr << ", iteration = " << minIndex << endl;

    cout << "done" << endl;
    return 0;

}
