#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>
#include <string>

#include "crbmComputer.h"

#include "../crbm.h"
using namespace std;
using namespace YAMATH;


CRBMStack::CRBMStack(int inLength, const char** inRBMFiles, int inGpuID)
{
    hipSetDevice(inGpuID);

    for(int i = 0; i < inLength; ++i)
    {
        CRBM::CRBMLayer *l = new CRBM::CRBMLayer();
        l->Load(inRBMFiles[i]);
        m_Layers.push_back(l);
    }
}

CRBMStack::CRBMStack(const std::vector<string> &inRBMFiles, int inGpuID)
{
    hipSetDevice(inGpuID);

    for(int i = 0; i < inRBMFiles.size(); ++i)
    {
        CRBM::CRBMLayer *l = new CRBM::CRBMLayer();
        l->Load(inRBMFiles[i]);
        m_Layers.push_back(l);
    }
}

CRBMStack::~CRBMStack(void)
{
    for(int i = 0; i < m_Layers.size(); ++i)
    {
        delete m_Layers[i];
        m_Layers[i] = NULL;
    }
}

void CRBMStack::Transform(int inLenInData, const float* inData, int inLenOutData, float* outData) const
{
    //MatrixCpu line(int inX = 1, int inY = 1, const float * inInit = NULL) //column first order
    MatrixCpu xCpu(1, inLenInData, inData);
    MatrixGpu xx = xCpu;
    MatrixGpu y;

    for(int i = 0; i < m_Layers.size(); ++i)
    {
        Timer t;
        //cout << "   Transforming with layer " << i+1 << endl;
        m_Layers[i]->Transform(xx, y);
        xx = y;
        cout << "Layer " << i << ": ";
        t.tac("");
    }

    MatrixCpu resx = xx;

    for(int i = 0; i < inLenOutData; ++i)
    {
        outData[i] = resx.getDataConst()[i];
    }
}

void CRBMStack::Transform(const std::vector<float> &inData, std::vector<float> &outData) const
{
    //MatrixCpu line(int inX = 1, int inY = 1, const float * inInit = NULL) //column first order
    MatrixCpu xCpu(1, inData.size(), &inData[0]);
    MatrixGpu xx = xCpu;
    MatrixGpu y;

    for(int i = 0; i < m_Layers.size(); ++i)
    {
        //cout << "   Transforming with layer " << i+1 << endl;
        m_Layers[i]->Transform(xx, y);
        xx = y;
    }

    MatrixCpu resx = xx;
    outData.assign(resx.getDataConst(), resx.getDataConst() + resx.getY());
}

int CRBMStack::GetOutputSize(void) const
{
    return  m_Layers.back()->getOutputSize();
}


//void RBMStack::Reconstruct(const std::vector<float> &inData, std::vector<float> &outData)
//{
//    //MatrixCpu line(int inX = 1, int inY = 1, const float * inInit = NULL) //column first order
//    MatrixCpu xx(1, iinData.size(), &inData[0])
//    MatrixGpu y;
//
//    for(int i = m_Weights.size() - 1; i >= 0; --i)
//    {
//        cout << "Reconstructing with weights " << i+1 << endl;
//        y = Mult(xx, (weights[i])->T());
//        xx = y;
//    }
//
//    MatrixCpu resx = xx;
//    outData.assign(resx.getDataConst(), resx.getDataConst() + resx.getY());
//}

