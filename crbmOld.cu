#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>
#include <string>
#include <time.h>

using namespace std;

/*
//#define CUDA

#ifdef CUDA
    //#include "deviceGPU.h"
    #include<hipblas.h>


#else
    #include "deviceCPU.h"
    typedef ComputingDevice::DeviceCPU dev;
#endif
*/

#include "matrix.h"

#include "crbm.h"


using namespace YAMATH;

typedef MatrixGpu Mat;

float computeError(Mat &inInp, Mat &inOut)
{
    Mat r2, r3;
    r2 = inInp - inOut;
    //msgG("in", inInp);
    //msgG("out", inOut);
    //msgG("r2", r2);
    r2 ^= 2.0f;
    r3 = r2.Sum();
    r3 *= 1.0f / inInp.getX();

    MatrixCpu rr = r3;

    msgG("abssum2", r3);

    return rr.getDataConst()[0];
}

int main(int argc, char** argv)
{
/*    if(argc != 7 && argc != 8)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " input-vector-file input-weight-file hidden-size learning-speed iter batch [cudadevice-id]" << endl;
        exit(1);
    }

    if(argc > 7)
    {
        int device = atoi(argv[7]);
        cout << "Device ID: " << device << endl;
        hipSetDevice(device);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    cout << "cublas init ..." << flush;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    cout << " done" << endl;

    int hidden = atoi(argv[3]);
    float lSpeed = atof(argv[4]);
    float iterations = atof(argv[5]);
    int batchSize = atoi(argv[6]);

    MatrixCpu *xCpu = new MatrixCpu();

    loadMatrix(*xCpu, argv[1]);

    int rows = xCpu->getX();
    int cols = xCpu->getY();

    Mat xx = *xCpu;
    msgG("loaded", xx);

    delete xCpu;
    xCpu = new MatrixCpu();

    if(string(argv[2]) != "-")
    {
        loadMatrix(*xCpu, string(argv[2]));
        msgC("w", *xCpu);
    }

//#define TEST
#ifdef TEST
    //image-size
    int im_x = 3;
    int im_y = 4;
    int im_z = 2;

    //convolution-size
    int im_cx = 2;
    int im_cy = 2;

    //stride-size
    int im_stridex = 1;
    int im_stridey = 1;
#else
    //image-size
    int im_x = 200;
    int im_y = 200;
    int im_z = 3;

    //convolution-size
    int im_cx = 10;
    int im_cy = 10;

    //stride-size
    int im_stridex = 5;
    int im_stridey = 5;
#endif
    CRBM::CRBMLayer abc(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, 15);

    Timer timer;

    int transX, transY;

    convolutionPatchesNumber(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, transX, transY);

    cout << "On image " << im_x << "x" << im_y << "x" << im_z << " applied convolution " << im_cx << "x" << im_cy << " with stride " << im_stridex << "x" << im_stridey << endl;
    cout << "It resulted into " << transX << "x" << transY << " patches." << endl;

    //int pn = transX*transY;

    Mat x, xraw, y, x2, y2, dw1, dw2, err, lastW;

    timer.tic();
    x = xx.Convolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey);
    timer.tac("Convolve: ");

    Mat w(x.getY(), hidden); //init weights
    cout << xCpu->getX() << ", " << xx.getX() << ", " << xCpu->getY() << ", " << hidden << endl;

    if(xCpu->getX() != w.getX() || xCpu->getY() != w.getY())
    {
        w.RandNormal(0.0f, 1.0f/(10*hidden));
        cout << "weight matrix randomized!" << endl;
    }
    else
    {
        w = *xCpu;
        cout << "weight matrix loaded!" << endl;
    }
    //msgG("w", w);
    delete xCpu;


    //w = 0.0f;
    ms("w", w);

    lastW = w;

    cout << endl;
    
    float minErr = FLT_MAX;
    int minIndex = 0;

    bool ONE_ROW = true;


    msgG("xxxxx", x);
    msgG("wwwww", w);

  
    for(int i = 0; i < iterations; ++i)
    {
        //Mat xraw = xx;
        //Mat xraw = xx.Sample(batchSize);

        //x = xraw.Convolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey);

        //cout << "x:" << x.getX() << ", y:" << x.getY() << endl;

        //saveGpuMatrix(x, string(argv[1]) + ".convolved");

        //Mat reverse;
        //reverse = x.DeConvolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, normalizer);

        //saveGpuMatrix(reverse, string(argv[1]) + ".reversed");
        //exit(1);

        y = Mult(x, w); // matrixwise -  y.shape = (dataA.x, weights.y) == (dataB.x, dataB.y)
        //msgG("y", y);
        //msgG("y=x*w", y);

        //y = y.Sigmoid();
        //msgG("y", y);

        x2 = Mult(y, w.T());
        //msgG("x2", x2);

        //x2 = x2.Sigmoid();
        //msgG("x2", x2);

        y2 = Mult(x2, w);
        //msgG("y2", y2);

        //y2 = y2.Sigmoid();
        //msgG("y2", y2);

        dw1 = Mult(x.T(), y);
        //msgG("dw1", dw1);
        dw2 = Mult(x2.T(), y2);
        //msgG("dw2", dw2);

        dw1 *= (lSpeed/x.getX());
        dw2 *= (lSpeed/x.getX());

        w = w + dw1;
        w = w - dw2;

        //lastW *= 0.00001;
        //w = w - lastW;

        lastW = w;
        //msgG("w", w);

        ms("w = w + dw", w);

        if(i % 50 == 0 || i+1 == iterations )
        {
            cout << i << ": ";
            float terr = computeError(x, x2);

            cout << "              " << flush;

            if(ONE_ROW)
            {
                cout << "              " << "\r" << flush;
            }
            else
            {
                cout << endl;
            }
            if(terr < minErr)
            {
                minErr = terr;
                minIndex = i;
            }
        }
    }
    cout << endl;

    MatrixCpu res = w;

    msgC("res", res);
    saveMatrix(res, string(argv[1]) + ".weights");

    y = Mult(x, w);
    //y = y.Sigmoid();
    //msgG("y - raw", y);
    //y.Reshape(xx.getX(), transX*transY*hidden);
    msgG("y", y);
    Mat yyy = y.TransformToUpperLayer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, xx.getX());
    msgG("trans(y)", yyy);
    MatrixCpu resy = yyy;
    saveMatrix(resy, string(argv[1]) + ".transform");
    Mat zzz = yyy.TransformFromUpperLayer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, xx.getX());
    msgG("retranst(trans(y))", zzz);

    //y.Reshape(hidden*xx.getX(), transX*transY);
    //msgG("reshaped(y)", y);
    //MatrixCpu resy = y;
    //saveMatrix(resy, string(argv[1]) + ".transformRaw");

    //exit(1);
    
    //Mat yy = y.T();
    //yy.MakeHardCopy();
    //msgG("transposed(y)", yy);
    //saveMatrix(resy, string(argv[1]) + ".transformRawTransposed");
    
    //saveMatrix(resy, string(argv[1]) + ".transform");
    exit(1);
    x2 = Mult(y, w.T());
    //x2 = x2.Sigmoid();
    Mat reverse, normalizer;
    timer.tic();
    normalizer = x.DeConvolveNormalizer(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, xx.getX());
    timer.tac("DeConvolveNormalizer: ");
    timer.tic();
    reverse = x2.DeConvolve(im_x, im_y, im_z, im_cx, im_cy, im_stridex, im_stridey, normalizer);
    timer.tac("DeConvolve: ");
    MatrixCpu resx = reverse;
    saveMatrix(resx, string(argv[1]) + ".reconstruct");


    cout << "done" << endl << "Min. test error = " << minErr << ", iteration = " << minIndex << endl;

    cout << "done" << endl;*/
    return 0;

}
