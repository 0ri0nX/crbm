#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>

using namespace std;

/*
//#define CUDA

#ifdef CUDA
    //#include "deviceGPU.h"
    #include<hipblas.h>


#else
    #include "deviceCPU.h"
    typedef ComputingDevice::DeviceCPU dev;
#endif
*/

#include "matrix.h"



using namespace YAMATH;

void loadMatrix(MatrixCpu &inM, char* filename, bool inTransposed = false)
{
    cout << "loading [" << filename << "] ... " << endl;
    ifstream f(filename);
    inM.Load(f, inTransposed);
    f.close();
}

void saveMatrix(MatrixCpu &inM, char* filename)
{
    cout << "saving [" << filename << "] ... " << endl;
    ofstream f(filename);
    inM.Save(f);
    f.close();
}

void msgG(char * inMsg, const MatrixGpu &inM)
{
    MatrixCpu x = inM;
    if(x.getX()*x.getY() > 400)
    {
        cout << "GPU: " << inMsg << ":" << endl;
        cout << x.getX() << " x " << x.getY() << endl;
        cout << "[ " << (x.getData()[0]) << " ... " << (x.getData()[x.getX()*x.getY()-1]) << " ]" << endl;
        cout << endl;
    }
    else if(x.getX()*x.getY() == 1)
    {
        cout << "GPU: " << inMsg << ":" << x.getData()[0] << flush;
    }
    else
    {
        cout << "GPU: " << inMsg << ":" << endl;
        x.Save(cout);
        cout << endl;
    }
}

void msgC(char * inMsg, const MatrixCpu &inM)
{
    cout << "CPU: " << inMsg << ":" << endl;
    const MatrixCpu &x = inM;
    if(x.getX()*x.getY() > 100)
    {
        cout << x.getX() << " x " << x.getY() << endl;
        cout << "[ " << (x.getDataConst()[0]) << " ... " << (x.getDataConst()[x.getX()*x.getY()-1]) << " ]" << endl;
    }
    else
    {
        x.Save(cout);
    }
    cout << endl;
}

void ms(char * inMsg, const MatrixGpu &inM)
{
    //msgG(inMsg, inM);
}


void testGpu(int x, int y)
{
    typedef MatrixGpu M;
    typedef MatrixCpu MC;
    
    cout << "GPU -----------" << endl;

    MC ac(x, y);
    for(int i = 0; i < ac.getX()*ac.getY(); ++i)
    {
        ac.getData()[i] = float(i);
    }
    M a = ac;
    msgG("a - init", a);

    //a = 11.0f;
    //msgG("a=11.0f", a);

    M b = a.AbsSum();
    msgG("b=a.AbsSum()", b);

    MC cc(y, 3);
    for(int i = 0; i < cc.getX()*cc.getY(); ++i)
    {
        cc.getData()[i] = 0.0f;
    }
    cc.getData()[0] = 1.0f;
    cc.getData()[y+1] = 1.0f;
    cc.getData()[2*y+2] = 1.0f;
    M c = cc;
    msgG("c", c);

    M d = a*c;
    msgG("d=a*c", d);

    
}

void testCpu(int x, int y)
{
    typedef MatrixCpu M;

    cout << "CPU -----------" << endl;
    
    M a(x, y);
    msgC("a - init", a);

    //a = 11.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        a.getData()[i] = 11;
    }
    msgC("a=11.0f", a);

    M b(1,1);
    float sum = 0.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        sum += a.getData()[i];
    }
    b.getData()[0] = sum;
    msgC("sum=a.AbsSum()", b);
}

//const float x1[] = {1.0f, 0.0f, 0.0f};
//const float t1[] = {1.0f, 0.0f};
//
//const float x2[] = {1.0f, 0.0f, 1.0f};
//const float t2[] = {1.0f, 0.0f};
//
//const float x3[] = {1.0f, 1.0f, 0.0f};
//const float t3[] = {1.0f, 0.0f};
//
//const float x4[] = {1.0f, 1.0f, 1.0f};
//const float t4[] = {0.0f, 1.0f};

typedef MatrixGpu Mat;

void computeError(Mat &inW, Mat &inInp, Mat &inOut)
{
    //cout << "inW:" << inW.getX() << " x " << inW.getY() << endl;
    //cout << "inInp:" << inInp.getX() << " x " << inInp.getY() << endl;
    Mat r, r2, r3;
    //msgG("www=", inW);
    //msgG("iiinp=", inInp);
    //msgG("ooout=", inOut);
    r = inInp * inW;
    //msgG("r=", r);
    r2 = r - inOut;
    //msgG("r2=", r2);
    r2 ^= 2.0f;
    //msgG("r2=", r2);
    r3 = r2.AbsSum();
    //msgG("r3=", r3);
    r3 *= 1.0f / inInp.getX();
    //msgG("r3=", r3);

    msgG("abssum2", r3);
}

int main(int argc, char** argv)
{
    if(argc != 5)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " input-vector-file target-vector-file output-weights-file learning-speed" << endl;
        exit(1);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    float lSpeed = atof(argv[4]);



    MatrixCpu *xxCpu = new MatrixCpu();
    MatrixCpu *ttCpu = new MatrixCpu();

    loadMatrix(*xxCpu, argv[1]);
    loadMatrix(*ttCpu, argv[2]);

    int rows = xxCpu->getX();
    int cols = xxCpu->getY();
    int colsT = ttCpu->getY();

    int fract = rows - rows/10;

    MatrixCpu *xCpu = new MatrixCpu(xxCpu->SubMatrix(0, 0, fract, cols));
    MatrixCpu *xCpuTe = new MatrixCpu(xxCpu->SubMatrix(fract, 0, rows, cols));

    MatrixCpu *tCpu = new MatrixCpu(ttCpu->SubMatrix(0, 0, fract, colsT));
    MatrixCpu *tCpuTe = new MatrixCpu(ttCpu->SubMatrix(fract, 0, rows, colsT));

    delete xxCpu;
    delete ttCpu;

    Mat x = *xCpu;
    Mat t = *tCpu;
    ms("x", x);
    ms("t", t);

    Mat xTe = *xCpuTe;
    Mat tTe = *tCpuTe;
    ms("x", xTe);
    ms("t", tTe);

    delete xCpu;
    delete tCpu;

    delete xCpuTe;
    delete tCpuTe;


    //Mat m;
    //m = x.AbsMax();
    //ms("absmax(x)", m);
    //m = x.AbsMin();
    //ms("absmin(x)", m);

    //m = t.AbsMax();
    //ms("absmax(t)", m);
    //m = t.AbsMin();
    //ms("absmin(t)", m);

    //x = MatrixCpu(1, 3, x1);
    //t = MatrixCpu(1, 2, t1);

    Mat w(x.getY(), t.getY()); //init weights
    w.Rand();
    w = 0.0f;
    ms("w", w);

    //w = x * t;

    //float alpha = 1.0f;
    //float beta = 0.0f;

    //hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    //        x.getX(), t.getY(), x.getY(),
    //        &alpha, x.getData(), x.getX(), t.getData(), t.getX(), &beta, w.getData(), w.getX());

    Mat y, e, suma, dw, dty;

    cout << endl;

    
    for(int i = 0; i < 2100; ++i)
    {
        //switch(i%4)
        //{
        //    case 0:
        //        x = MatrixCpu(1, 3, x1);
        //        t = MatrixCpu(1, 2, t1);
        //        break;
        //    case 1:
        //        x = MatrixCpu(1, 3, x2);
        //        t = MatrixCpu(1, 2, t2);
        //        break;
        //    case 2:
        //        x = MatrixCpu(1, 3, x3);
        //        t = MatrixCpu(1, 2, t3);
        //        break;
        //    case 3:
        //        x = MatrixCpu(1, 3, x4);
        //        t = MatrixCpu(1, 2, t4);
        //        break;
        //    default:
        //        exit(0);
        //}
        //ms("x", x);
        //ms("t", t);
        y = x * w; // matrixwise -  y.shape = (dataA.x, weights.y) == (dataB.x, dataB.y)
        ms("y=x*w", y);

        dty = t - y;
        ms("dty=t-y", dty);

        if(i % 1 == 0)
        {
            //e = dty;
            ////ms("e=dty", e);
    
            //e ^= 2.0f;//elementwise
            ////ms("e^=2", e);
    
            cout /*<< "\r"*/ << i << ": ";
            //suma = e.AbsSum();
            //suma *= 1.0f / x.getX();
            //msgG("abssum", suma);
            //msgG("x", x);
            //msgG("w", w);
            computeError(w, x, t);
            computeError(w, xTe, tTe);
            cout << endl;
            //cout << "error:" << ee << endl;
            //if(ee < 0.001f)
            //{
            //    break;
            //}

        }

        dw = (x^"T") * dty;
        //ms("dty", dty);
        ms("dw=x^t * dty", dw);

        //dw*= 0.001f * 1.0f/(x.getX()*x.getY());
        //ms("dw*= 000.1 * 1.0f/(x.getX()*x.getY())", dw);

        dw*= lSpeed;
        ms("dw*= lSpeed", dw);

        w = w + dw;
        ms("w = w + dw", w);

        
        //Mat y = x * w; // matrixwise -  y.shape = (dataA.x, weights.y) == (dataB.x, dataB.y)
        //dev::Matrix e = 0.5f*(t - y)^2; //yDiff.shape = dataB.shape

        //if(i % 10 == 0)
        //{
        //    float ee = dev::sumSquared(e); // ee = sum(e^2) elementwise squared sum
        //    cout << "error:" << ee << endl;
        //    if(ee < 0.001f)
        //    {
        //        break;
        //    }
        //}

        ////(t - y)dFi*xi => (t - y)*xi
        //dev::Matrix dW = x.trans * e; // == (y - dataB)*dataA ; // elementwise
        //
        //w += alpha*dW;
        //cout << "]" << endl; 
    }    

    MatrixCpu res = w;

    msgC("res=", res);
    saveMatrix(res, argv[3]);
    //res.Save(cout);

    cout << "done" << endl;
    return 0;

}
