#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>

using namespace std;

/*
//#define CUDA

#ifdef CUDA
    //#include "deviceGPU.h"
    #include<hipblas.h>


#else
    #include "deviceCPU.h"
    typedef ComputingDevice::DeviceCPU dev;
#endif
*/

#include "matrix.h"



using namespace YAMATH;

void loadMatrix(MatrixCpu &inM, char* filename, bool inTransposed = false)
{
    cout << "loading [" << filename << "] ... " << endl;
    ifstream f(filename);
    inM.Load(f, inTransposed);
    f.close();
}

void saveMatrix(MatrixCpu &inM, char* filename)
{
    cout << "saving [" << filename << "] ... " << endl;
    ofstream f(filename);
    inM.Save(f);
    f.close();
}

void msgC(char * inMsg, const MatrixCpu &x)
{
    int n = x.getX()*x.getY();
    if(n > 400)
    {
        cout << inMsg << ": " << x.getX() << " x " << x.getY()
             << "[ " << (x.getDataConst()[0]) << ", " << (x.getDataConst()[1]) << " ... " << (x.getDataConst()[n-2]) << ", " << (x.getDataConst()[n-1]) << " ]" << endl;
    }
    else if(n == 1)
    {
        cout  << inMsg << ":[" << x.getDataConst()[0] << "]" << flush;
    }
    else
    {
        cout  << inMsg << ":" << endl;
        x.Save(cout);
        cout << endl;
    }
}

void msgG(char * inMsg, const MatrixGpu &inM)
{
    MatrixCpu x = inM;
    msgC(inMsg, x);
}

void ms(char * inMsg, const MatrixGpu &inM)
{
    //msgG(inMsg, inM);
}


void testGpu(int x, int y)
{
    typedef MatrixGpu M;
    typedef MatrixCpu MC;
    
    cout << "GPU -----------" << endl;

    MC ac(x, y);
    for(int i = 0; i < ac.getX()*ac.getY(); ++i)
    {
        ac.getData()[i] = float(i);
    }
    M a = ac;
    msgG("a - init", a);

    //a = 11.0f;
    //msgG("a=11.0f", a);

    M b = a.AbsSum();
    msgG("b=a.AbsSum()", b);

    MC cc(y, 3);
    for(int i = 0; i < cc.getX()*cc.getY(); ++i)
    {
        cc.getData()[i] = 0.0f;
    }
    cc.getData()[0] = 1.0f;
    cc.getData()[y+1] = 1.0f;
    cc.getData()[2*y+2] = 1.0f;
    M c = cc;
    msgG("c", c);

    M d = Mult(a, c);
    msgG("d=a*c", d);

    
}

void testCpu(int x, int y)
{
    typedef MatrixCpu M;

    cout << "CPU -----------" << endl;
    
    M a(x, y);
    msgC("a - init", a);

    //a = 11.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        a.getData()[i] = 11;
    }
    msgC("a=11.0f", a);

    M b(1,1);
    float sum = 0.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        sum += a.getData()[i];
    }
    b.getData()[0] = sum;
    msgC("sum=a.AbsSum()", b);
}

//const float x1[] = {1.0f, 0.0f, 0.0f};
//const float t1[] = {1.0f, 0.0f};
//
//const float x2[] = {1.0f, 0.0f, 1.0f};
//const float t2[] = {1.0f, 0.0f};
//
//const float x3[] = {1.0f, 1.0f, 0.0f};
//const float t3[] = {1.0f, 0.0f};
//
//const float x4[] = {1.0f, 1.0f, 1.0f};
//const float t4[] = {0.0f, 1.0f};

typedef MatrixGpu Mat;

void computeError(Mat &inW, Mat &inInp, Mat &inOut)
{
    //cout << "inW:" << inW.getX() << " x " << inW.getY() << endl;
    //cout << "inInp:" << inInp.getX() << " x " << inInp.getY() << endl;
    Mat r, r2, r3;
    //msgG("www=", inW);
    //msgG("iiinp=", inInp);
    //msgG("ooout=", inOut);
    r = Mult(inInp, inW);
    //msgG("r=", r);
    r2 = r - inOut;
    //msgG("r2=", r2);
    r2 ^= 2.0f;
    //msgG("r2=", r2);
    r3 = r2.AbsSum();
    //msgG("r3=", r3);
    r3 *= 1.0f / inInp.getX();
    //msgG("r3=", r3);

    msgG("abssum2", r3);
}

int main(int argc, char** argv)
{
    if(argc != 5)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " input-vector-file target-vector-file output-weights-file learning-speed" << endl;
        exit(1);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    float lSpeed = atof(argv[4]);

    MatrixCpu *xxCpu = new MatrixCpu();
    MatrixCpu *ttCpu = new MatrixCpu();

    loadMatrix(*xxCpu, argv[1]);
    msgC("matrix: ", *xxCpu);
    loadMatrix(*ttCpu, argv[2]);
    msgC("matrix: ", *ttCpu);

    int rows = xxCpu->getX();
    int cols = xxCpu->getY();
    int colsT = ttCpu->getY();

    int fract = rows - rows/5;

    MatrixCpu *xCpu = new MatrixCpu(xxCpu->SubMatrix(0, 0, fract, cols));
    MatrixCpu *xCpuTe = new MatrixCpu(xxCpu->SubMatrix(fract, 0, rows, cols));

    MatrixCpu *tCpu = new MatrixCpu(ttCpu->SubMatrix(0, 0, fract, colsT));
    MatrixCpu *tCpuTe = new MatrixCpu(ttCpu->SubMatrix(fract, 0, rows, colsT));

    delete xxCpu;
    delete ttCpu;

    Mat x = *xCpu;
    Mat t = *tCpu;

    Mat xTe = *xCpuTe;
    Mat tTe = *tCpuTe;

    delete xCpu;
    delete tCpu;

    delete xCpuTe;
    delete tCpuTe;


    Mat w(x.getY(), t.getY()); //init weights
    //w.Rand();
    w = 0.0f;

    //learning speed matrix
    Mat ls(x.getY(), t.getY());
    ls = lSpeed;

    Mat lastDir(x.getY(), t.getY());
    lastDir = 0.0f;

    Mat y, e, suma, dw, dty, lsModUp, lsModDown, actDir, lsModMin;

    cout << endl;

    
    for(int i = 0; i < 100000; ++i)
    {
        y = Mult(x, w); // matrixwise -  y.shape = (dataA.x, weights.y) == (dataB.x, dataB.y)
        //msgG("y=x*w", y);

        dty = t - y;
        //msgG("dty=t-y", dty);

        if(i % 1 == 0)
        {
            cout /*<< "\r"*/ << i << ": ";
            computeError(w, x, t);
            computeError(w, xTe, tTe);
            cout << endl;
        }

        dw = Mult(x.T(), dty);
        ms("dw=x^t * dty", dw);

        actDir = dw;
        lsModUp = Mat(lastDir*actDir) >= 0.0f;
        lsModDown = lsModUp <= 0.0f;
        lsModMin = ls < (lSpeed*0.0001f);
        lsModUp *= 1.1f;
        lsModDown *= 0.5f;
        lsModMin *= lSpeed*0.0001f;

        //msgG("dir up      ", lsModUp);
        //msgG("dir down    ", lsModDown);

        ls = ls * Mat(lsModUp + lsModDown);
        //ls = ls * lsMod;//Mat(Mat(Mat(lsMod>=0)*1.1f) + Mat(lsMod<0)*0.5f);
        ls = ls + lsModMin;
        //msgG("speed matrix", ls);

        lastDir = actDir;
        
        dw = dw * ls;
        ms("dw*= lSpeed", dw);

        w = w + dw;
        ms("w = w + dw", w);
    }    

    MatrixCpu res = w;

    msgC("res=", res);
    saveMatrix(res, argv[3]);

    cout << "done" << endl;
    return 0;

}
