#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>
#include <string>
#include <time.h>
#include <csignal>
#include <iomanip>
#include <cmath>

using namespace std;

#include "matrix.h"
#include "utils.h"
#include "crbm.h"

using namespace YAMATH;

typedef MatrixGpu Mat;

string getName(const string &inPrefix, int inIdx, int inTotal)
{
    stringstream s;
    int w = ceil(log10(inTotal));
    s << inPrefix.c_str() << setfill('0') << setw(w) << inIdx;

    return s.str();
}

int main(int argc, char** argv)
{
    if(argc < 5)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " <gpu-id> <reconstruct|transform> input-vector-file crbm-file1 [crbm-file2] ..." << endl;
        exit(1);
    }


    hipSetDevice(atoi(argv[1]));
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    string computationType = argv[2];
    if(computationType != "reconstruct" && computationType != "transform")
    {
        cout << "Unsupported computation type: [" << computationType << "]" << endl;
        exit(1);
    }

    int batchSize = 500;

    cout << "Maximal batch size: " << batchSize << endl;

    MatrixCpu *xCpu = new MatrixCpu();

    loadMatrix(*xCpu, argv[3]);//, false, string(argv[3]) + ".cache");

    int rows = xCpu->getX();
    int cols = xCpu->getY();

    int batchNum = (rows - 1) / batchSize + 1;

    std::vector<CRBM::CRBMLayer*> layers;

    for(int i = 4; i < argc; ++i)
    {
        cout << i-3 << ". ";
        CRBM::CRBMLayer *l = new CRBM::CRBMLayer();
        l->Load(string(argv[i]));
        layers.push_back(l);
    }

    int resSize = -1;
    string outFilename = string(argv[3]);

    if(computationType == "transform")
    {
        outFilename += ".transformed";

        int outX, outY;
        layers.back()->getConvolutionPatchesNumber(outX, outY);
        resSize = outX*outY*layers.back()->s().hidden;
    }
    else
    {
        outFilename += ".reconstruct";
        resSize = cols;
    }

    cout << "Saving into: [" << outFilename << "]" << endl;

    ofstream f(outFilename.c_str());

    const int saveVersion = 0;

    MatrixCpu::SaveHeader(f, rows, resSize, saveVersion);
    //f << rows << " " << resSize << endl;

    Mat xx;
    MatrixCpu tmpxx;
    Timer timer;

    for(int batch = 0; batch < batchNum; ++batch)
    {
        int a = batch*batchSize;
        int b = min((batch+1)*batchSize, rows);

        cout << batch+1 << " / " << batchNum << endl;

        timer.tic();
        xx = xCpu->SubMatrix(a, 0, b, cols);
        timer.tac("   selected: ");

        //msgG("loaded", xx);

        Mat y;
        //msgG("xx", xx);
        //layers[0]->Convolve(xx, y);
        //saveMatrix(y, string(argv[3]) + ".conv");
        //msgG("conv(xx)", y);
        //layers[0]->DeConvolve(y, xx);
        //saveMatrix(xx, string(argv[3]) + ".convDeconv");
        //msgG("deconv(conv(xx))", xx);

        //exit(1);

        for(int i = 0; i < layers.size(); ++i)
        {
            timer.tic();
            cout << "   Transforming with layer " << i+1 << flush;
            layers[i]->Transform(xx, y);
            timer.tac(" ");
            xx = y;
        }

        if(computationType == "transform")
        {
            timer.tic();
            tmpxx = xx;
            tmpxx.Save(f, false, saveVersion);
            timer.tac("   saved: ");
            continue;
        }
        
        for(int i = layers.size() - 1; i >= 0; --i)
        {
            timer.tic();
            cout << "   Reconstructing with layer " << i+1 << flush;
            layers[i]->Reconstruct(xx, y);
            timer.tac(" ");
            xx = y;
        }

        if(computationType == "reconstruct")
        {
            timer.tic();
            tmpxx = xx;
            tmpxx.Save(f, false, saveVersion);
            timer.tac("   saved: ");
            continue;
        }
    }

    f.close();
    cout << "Saved into: [" << outFilename << "]" << endl;

    return 0;
}
