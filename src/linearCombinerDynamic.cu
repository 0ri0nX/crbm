#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cassert>

using namespace std;

/*
//#define CUDA

#ifdef CUDA
    //#include "deviceGPU.h"
    #include<hipblas.h>


#else
    #include "deviceCPU.h"
    typedef ComputingDevice::DeviceCPU dev;
#endif
*/

#include "matrix.h"



using namespace YAMATH;

void loadMatrix(MatrixCpu &inM, char* filename, bool inTransposed = false)
{
    cout << "loading [" << filename << "] ... " << endl;
    ifstream f(filename);
    inM.Load(f, inTransposed);
    f.close();
}

void saveMatrix(MatrixCpu &inM, char* filename)
{
    cout << "saving [" << filename << "] ... " << endl;
    ofstream f(filename);
    inM.Save(f);
    f.close();
}

void msgS(char * inMsg, const MatrixGpu &x)
{
    cout << " " << inMsg << ": " << x.getX() << " x " << x.getY();
}

void msgC(char * inMsg, const MatrixCpu &x)
{
    int n = x.getX()*x.getY();
    if(n > 400)
    {
        cout << inMsg << ": " << x.getX() << " x " << x.getY()
             << "[ " << (x.getDataConst()[0]) << ", " << (x.getDataConst()[1]) << " ... " << (x.getDataConst()[n-2]) << ", " << (x.getDataConst()[n-1]) << " ]" << flush;
    }
    else if(n == 1)
    {
        cout  << inMsg << ":[" << x.getDataConst()[0] << "]" << flush;
    }
    else
    {
        cout  << inMsg << ":" << endl;
        x.Save(cout);
        cout << endl;
    }
}

void msgG(char * inMsg, const MatrixGpu &inM)
{
    MatrixCpu x = inM;
    msgC(inMsg, x);
}

void ms(char * inMsg, const MatrixGpu &inM)
{
    //msgG(inMsg, inM);
}


void testGpu(int x, int y)
{
    typedef MatrixGpu M;
    typedef MatrixCpu MC;
    
    cout << "GPU -----------" << endl;

    MC ac(x, y);
    for(int i = 0; i < ac.getX()*ac.getY(); ++i)
    {
        ac.getData()[i] = float(i);
    }
    M a = ac;
    msgG("a - init", a);

    //a = 11.0f;
    //msgG("a=11.0f", a);

    M b = a.AbsSum();
    msgG("b=a.AbsSum()", b);

    MC cc(y, 3);
    for(int i = 0; i < cc.getX()*cc.getY(); ++i)
    {
        cc.getData()[i] = 0.0f;
    }
    cc.getData()[0] = 1.0f;
    cc.getData()[y+1] = 1.0f;
    cc.getData()[2*y+2] = 1.0f;
    M c = cc;
    msgG("c", c);

    M d = Mult(a, c);
    msgG("d=a*c", d);

    
}

void testCpu(int x, int y)
{
    typedef MatrixCpu M;

    cout << "CPU -----------" << endl;
    
    M a(x, y);
    msgC("a - init", a);

    //a = 11.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        a.getData()[i] = 11;
    }
    msgC("a=11.0f", a);

    M b(1,1);
    float sum = 0.0f;
    for(int i = 0; i < a.getX()*a.getY(); ++i)
    {
        sum += a.getData()[i];
    }
    b.getData()[0] = sum;
    msgC("sum=a.AbsSum()", b);
}

//const float x1[] = {1.0f, 0.0f, 0.0f};
//const float t1[] = {1.0f, 0.0f};
//
//const float x2[] = {1.0f, 0.0f, 1.0f};
//const float t2[] = {1.0f, 0.0f};
//
//const float x3[] = {1.0f, 1.0f, 0.0f};
//const float t3[] = {1.0f, 0.0f};
//
//const float x4[] = {1.0f, 1.0f, 1.0f};
//const float t4[] = {0.0f, 1.0f};

typedef MatrixGpu Mat;

float computeError(Mat &inW, Mat &inInp, Mat &inOut)
{
    Mat r, r2, r3;
    r = Mult(inInp, inW);
    r2 = r - inOut;
    r2 ^= 2.0f;
    r3 = r2.Sum();
    r3 *= 1.0f / inInp.getX();

    MatrixCpu rr = r3;

    //msgG("abssum2", r3);

    return rr.getDataConst()[0];
}

__global__ void testKernel1(float *lSpeed, const float* lastDir, const float* actDir, float minSpeed, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n)
    {
        bool goodDir = lastDir[i]*actDir[i] >= 0;

        lSpeed[i] *= goodDir ? 1.1f : 0.5f;

        if(lSpeed[i] < minSpeed)
        {
            lSpeed[i] = minSpeed;
        }
    }
}

__global__ void testKernel2(float *lSpeed, const float* lastDir, const float* actDir, float minSpeed, float maxSpeed, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n)
    {
        float goodDir = int(lastDir[i]*actDir[i] >= 0);

        float speed = lSpeed[i];

        speed *= goodDir*1.1f + (goodDir-1.0f)*0.5f;

        if(speed < minSpeed)
        {
            speed = minSpeed;
        }

        if(speed > maxSpeed)
        {
            speed = maxSpeed;
        }

        lSpeed[i] = speed;
    }
}

//ls = ls * Mat(Mat(Mat(Mat(lastDir*actDir)>=0)*1.1f) + Mat(Mat(lastDir*actDir)<0)*0.5f);
void testKernelCall(Mat &ls, const Mat &ld, const Mat &ad, float speed)
{
    int n = ls.getX()*ls.getY();
    int tpb = 512;
    int b = (n-1) / tpb + 1;

    float minSpeed = speed*0.01;
    float maxSpeed = speed*100;

    testKernel2<<<b, tpb>>>(ls.getData(), ld.getDataConst(), ad.getDataConst(), minSpeed, maxSpeed, n);
}


int main(int argc, char** argv)
{
    if(argc != 7)
    {
        cout << "Too few params!" << endl;
        cout << argv[0] << " input-vector-file target-vector-file output-weights-file learning-speed iteri GpuId" << endl;
        exit(1);
    }

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    hipSetDevice(atoi(argv[6]));

    float lSpeed = atof(argv[4]);
    float iterations = atof(argv[5]);

    MatrixCpu *xxCpu = new MatrixCpu();
    MatrixCpu *ttCpu = new MatrixCpu();

    loadMatrix(*xxCpu, argv[1]);
    msgC("matrix: ", *xxCpu); cout << endl;
    loadMatrix(*ttCpu, argv[2]);
    msgC("matrix: ", *ttCpu); cout << endl;

    int rows = xxCpu->getX();
    int cols = xxCpu->getY();
    int colsT = ttCpu->getY();

    int fract = rows - rows/5;

    MatrixCpu *xCpu = new MatrixCpu(xxCpu->SubMatrix(0, 0, fract, cols));
    MatrixCpu *xCpuTe = new MatrixCpu(xxCpu->SubMatrix(fract, 0, rows, cols));

    MatrixCpu *tCpu = new MatrixCpu(ttCpu->SubMatrix(0, 0, fract, colsT));
    MatrixCpu *tCpuTe = new MatrixCpu(ttCpu->SubMatrix(fract, 0, rows, colsT));

    delete xxCpu;
    delete ttCpu;

    Mat x = *xCpu;
    Mat t = *tCpu;

    Mat xTe = *xCpuTe;
    Mat tTe = *tCpuTe;

    delete xCpu;
    delete tCpu;

    delete xCpuTe;
    delete tCpuTe;


    Mat w(x.getY(), t.getY()); //init weights
    //w.RandNormal(0.0, 0.0001);
    w = 0.0f;

    //learning speed matrix
    Mat ls(x.getY(), t.getY());
    ls = lSpeed;

    Mat lastDir(x.getY(), t.getY());
    lastDir = 0.0f;

    Mat y, e, suma, dw, dty, actDir;
    //Mat lsModUp, lsModDown, lsModMin
    cout << endl;

    float minErr = FLT_MAX;
    int minIndex = 0;

    bool ONE_ROW = true;
    
    for(int i = 0; i < iterations; ++i)
    {
        y = Mult(x, w); // matrixwise -  y.shape = (dataA.x, weights.y) == (dataB.x, dataB.y)
        //msgG("y=x*w", y);

        dty = t - y;
        //msgG("dty=t-y", dty);

        dw = Mult(x.T(), dty);
        dw *= 1.0 / x.getX();
        ms("dw=x^t * dty", dw);

        actDir = dw;

        //dynamic learning speed using matrix library
        if(0)
        {
            Mat lsModUp = Mat(lastDir*actDir) >= 0.0f;
            Mat lsModDown = lsModUp <= 0.0f;
            Mat lsModMin = ls < (lSpeed*0.0001f);
            lsModUp *= 1.01f;
            lsModDown *= 0.5f;
            lsModMin *= lSpeed*0.0001f;

            //msgG("dir up      ", lsModUp);
            //msgG("dir down    ", lsModDown);

            ls = ls * Mat(lsModUp + lsModDown);
            //ls = ls * Mat(Mat(Mat(Mat(lastDir*actDir)>=0)*1.1f) + Mat(Mat(lastDir*actDir)<0)*0.5f);
            ls = ls + lsModMin;
        }

        //dynamic learning speed using specialised kernel - faster by 20%
        if(1)
        {
            testKernelCall(ls, lastDir, actDir, lSpeed);
        }
        //msgG("speed matrix", ls);

        lastDir = actDir;
        
        dw = dw * ls;
        ms("dw*= lSpeed", dw);

        w = w + dw;
        ms("w = w + dw", w);

        if(i % 100 == 0 || i+1 == iterations )
        {
            cout << i << ": ";
            //computeError(w, x, t);
            float terr = computeError(w, xTe, tTe);
            cout << terr;
            Mat mx = ls.Min();
            msgG(", min learning coef", mx);
            mx = ls.Max();
            msgG(", max learning coef", mx);
            msgG("", ls);
            if(ONE_ROW)
            {
                cout << "              " << "\r" << flush;
            }
            else
            {
                cout << endl;
            }

            if(terr < minErr)
            {
                minErr = terr;
                minIndex = i;
            }
        }

       //msgS("y", y);msgS("e", e);msgS("suma", suma);msgS("dw", dw);msgS("dty", dty);msgS("lsModUp", lsModUp);msgS("lsModDown", lsModDown);msgS("actDir", actDir);msgS("lsModMin", lsModMin);
    }
    cout << endl; 

    MatrixCpu res = w;

    msgC("res", res);
    saveMatrix(res, argv[3]);

    cout << "done" << endl << "Min. test error = " << minErr << ", iteration = " << minIndex << endl;
    return 0;

}
